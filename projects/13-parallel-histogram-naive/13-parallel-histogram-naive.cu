
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <random>
#include <chrono>

#define CUDA_CHECK(err)                                                                          \
  {                                                                                              \
    if (err != hipSuccess)                                                                      \
    {                                                                                            \
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }

constexpr int N = 512 * 512 * 512;
constexpr char initial_char = 'A';
constexpr int num_bins = 64;
// Cannot be too large due because of signed char
// ranging from -128 to 127 (A is 65)
constexpr int num_unique_chars = 50;

constexpr int threads_per_block = 256;
constexpr int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

/*
 * Computes a histogram of character frequencies.
 * - Each thread processes one input character.
 * - Race conditions are prevented by using atomicAdd().
 */
__global__ void histogram_kernel(const char *d_input_string, unsigned int *d_histogram)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N)
  {
    char current_char = d_input_string[idx];
    unsigned int bin_index = (current_char - initial_char) * num_bins / num_unique_chars;
    atomicAdd(&d_histogram[bin_index], 1);
  }
}

int verify_histogram(const char *h_input_string, const unsigned int *h_histogram)
{
  unsigned int target_histogram[num_bins] = {0};

  for (int i = 0; i < N; i++)
  {
    char current_char = h_input_string[i];
    unsigned int bin_index = (current_char - initial_char) * num_bins / num_unique_chars;
    target_histogram[bin_index]++;
  }

  for (int i = 0; i < num_bins; i++)
  {
    if (target_histogram[i] != h_histogram[i])
    {
      printf("Mismatch at bin %d: expected %u, got %u\n", i, target_histogram[i], h_histogram[i]);
      return 1;
    }
  }

  return 0;
}

int main()
{
  // Define input string
  size_t input_memsize = N * sizeof(char);
  std::vector<char> h_input_string(N);

  // Make 90% characters the same to make the performance
  // roughly 3-4 times worse, use all characters for the last 10%
  for (int i = 0; i < N; i++)
  {
    if (i < 0.9 * N)
    {
      h_input_string[i] = initial_char;
    }
    else
    {
      h_input_string[i] = initial_char + (i % (num_unique_chars - 1)) + 1;
    }
  }

  // Allocate memory for the host histogram result
  size_t histogram_memsize = num_bins * sizeof(unsigned int);
  std::vector<unsigned int> h_histogram(num_bins);

  // Prepare device variables
  char *d_input_string;
  unsigned int *d_histogram;
  CUDA_CHECK(hipMalloc((void **)&d_input_string, input_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_histogram, histogram_memsize));

  // Move data from host to device
  CUDA_CHECK(hipMemcpy(d_input_string, h_input_string.data(), input_memsize, hipMemcpyHostToDevice));

  // Initialize the histogram memory on the device to all zeros
  CUDA_CHECK(hipMemset(d_histogram, 0, histogram_memsize));

  // Create events for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  // Perform histogram calculation on GPU
  histogram_kernel<<<blocks_per_grid, threads_per_block>>>(d_input_string, d_histogram);
  CUDA_CHECK(hipGetLastError());

  // Record the end time and synchronize
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  // Calculate elapsed time
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Kernel execution time: %f ms (use NCU for a more precise measurement!)\n", milliseconds);

  // Move data from device to host
  CUDA_CHECK(hipMemcpy(h_histogram.data(), d_histogram, histogram_memsize, hipMemcpyDeviceToHost));

  // Check values
  printf("Verifying histogram...\n");
  if (verify_histogram(h_input_string.data(), h_histogram.data()) != 0)
  {
    return 1;
  }
  printf("All values match\n");

  // Free memory and destroy events
  CUDA_CHECK(hipFree(d_input_string));
  CUDA_CHECK(hipFree(d_histogram));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}