#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <random>
#include <chrono>
#include <queue>
#include "graph/graph.h"

#define CUDA_CHECK(err)                                                                          \
  {                                                                                              \
    if (err != hipSuccess)                                                                      \
    {                                                                                            \
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }

constexpr int num_vertices = 128 * 1024;
constexpr int num_edges = 1024 * 1024;

constexpr int threads_per_block = 1024;
constexpr int local_frontier_capacity = 4096;

/*
 * Performs BFS for a given level.
 - Iterates over the previous frontier and adds unvisited vertices to the current frontier
   while saving their distance.
 - Uses shared memory to construct the part of the current frontier for the given block,
   commits it to the global frontier at the end.
 - The shared frontier depending on <local_frontier_capacity>
   needs to be able to fit into the shared memory.
 - Each thread is responsible for one vertex from the previous frontier.
 */
__global__ void bfs_kernel(const int *row_ptr, const int *col_indices, int *dist,
                           int *prev_frontier, int *prev_frontier_num,
                           int *curr_frontier, int *curr_frontier_num,
                           const int prev_level)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ int shared_curr_frontier[local_frontier_capacity];
  __shared__ int shared_curr_frontier_num;

  if (threadIdx.x == 0)
  {
    shared_curr_frontier_num = 0;
  }

  __syncthreads();

  if (i < *prev_frontier_num)
  {
    int u = prev_frontier[i];

    for (int j = row_ptr[u]; j < row_ptr[u + 1]; j++)
    {
      int v = col_indices[j];

      if (atomicCAS(&dist[v], -1, prev_level + 1) == -1)
      {
        int shared_idx = atomicAdd(&shared_curr_frontier_num, 1);
        if (shared_idx < local_frontier_capacity)
        {
          shared_curr_frontier[shared_idx] = v;
        }
        else
        {
          int global_idx = atomicAdd(curr_frontier_num, 1);
          curr_frontier[global_idx] = v;
        }
      }
    }
  }

  __syncthreads();

  int items_to_copy = min(shared_curr_frontier_num, local_frontier_capacity);

  __shared__ int curr_frontier_start;
  if (threadIdx.x == 0)
  {
    curr_frontier_start = atomicAdd(curr_frontier_num, items_to_copy);
  }

  __syncthreads();

  for (int i = threadIdx.x; i < items_to_copy; i += blockDim.x)
  {
    curr_frontier[curr_frontier_start + i] = shared_curr_frontier[i];
  }
}

int main()
{
  // Create random number generator and random distribution
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);

  // Define host graph by first constructing a spanning tree
  // then adding random edges up to <num_edges>
  std::vector<Edge> edges = generate_random_spanning_tree(num_vertices, generator);
  add_random_edges(edges, num_vertices, num_edges, generator);

  // Convert the graph to the CSR format
  CSRGraph graph = convert_to_csr(edges, num_vertices);

  // Allocate memory for host output vector and initialize to -1,
  // set distance for the source node to 0
  std::vector<int> h_dist(num_vertices);
  std::fill(h_dist.begin(), h_dist.end(), -1);
  int source_vertex = 0;
  h_dist[source_vertex] = 0;

  // Prepare device variables
  size_t row_ptr_memsize = graph.row_ptr.size() * sizeof(int);
  size_t col_indices_memsize = graph.col_indices.size() * sizeof(int);
  size_t dist_memsize = graph.num_vertices * sizeof(int);
  size_t prev_frontier_memsize = graph.num_vertices * sizeof(int);
  size_t prev_frontier_num_memsize = sizeof(int);
  size_t curr_frontier_memsize = graph.num_vertices * sizeof(int);
  size_t curr_frontier_num_memsize = sizeof(int);

  int *d_row_ptr, *d_col_indices, *d_dist,
      *d_prev_frontier, *d_prev_frontier_num,
      *d_curr_frontier, *d_curr_frontier_num;

  CUDA_CHECK(hipMalloc((void **)&d_row_ptr, row_ptr_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_col_indices, col_indices_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_dist, dist_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_prev_frontier, prev_frontier_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_prev_frontier_num, prev_frontier_num_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_curr_frontier, curr_frontier_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_curr_frontier_num, curr_frontier_num_memsize));

  // Copy data from host to device
  CUDA_CHECK(hipMemcpy(d_row_ptr, graph.row_ptr.data(), row_ptr_memsize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_col_indices, graph.col_indices.data(), col_indices_memsize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_dist, h_dist.data(), dist_memsize, hipMemcpyHostToDevice));

  // Set previous frontier
  int h_prev_frontier_num = 1;
  CUDA_CHECK(hipMemcpy(d_prev_frontier_num, &h_prev_frontier_num, sizeof(int), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_prev_frontier, &source_vertex, sizeof(int), hipMemcpyHostToDevice));

  // Create events for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  // Run BFS and compute distances on GPU
  int prev_level = 0;
  while (h_prev_frontier_num > 0)
  {
    int h_curr_frontier_num = 0;
    CUDA_CHECK(hipMemcpy(d_curr_frontier_num, &h_curr_frontier_num, sizeof(int), hipMemcpyHostToDevice));

    int blocks_per_grid = (h_prev_frontier_num + threads_per_block - 1) / threads_per_block;
    bfs_kernel<<<blocks_per_grid, threads_per_block>>>(d_row_ptr, d_col_indices, d_dist,
                                                       d_prev_frontier, d_prev_frontier_num,
                                                       d_curr_frontier, d_curr_frontier_num,
                                                       prev_level);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(&h_curr_frontier_num, d_curr_frontier_num, sizeof(int), hipMemcpyDeviceToHost));

    int *temp_ptr_frontier = d_prev_frontier;
    d_prev_frontier = d_curr_frontier;
    d_curr_frontier = temp_ptr_frontier;

    int *temp_ptr_frontier_num = d_prev_frontier_num;
    d_prev_frontier_num = d_curr_frontier_num;
    d_curr_frontier_num = temp_ptr_frontier_num;

    h_prev_frontier_num = h_curr_frontier_num;

    prev_level++;
  }

  // Record the end time and synchronize
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  // Calculate elapsed time
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Kernel execution time: %f ms (use NCU for a more precise measurement!)\n", milliseconds);

  // Copy data from device to host
  CUDA_CHECK(hipMemcpy(h_dist.data(), d_dist, dist_memsize, hipMemcpyDeviceToHost));

  // Verify the result and measure CPU execution time
  auto cpu_start = std::chrono::high_resolution_clock::now();
  if (verify_bfs(graph, h_dist, source_vertex) != 0)
  {
    printf("Verification failed\n");
  }
  else
  {
    printf("All values match\n");
  }
  auto cpu_end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> cpu_time = cpu_end - cpu_start;
  printf("CPU execution time: %f milliseconds\n", cpu_time.count());

  // Free memory and destroy events
  CUDA_CHECK(hipFree(d_row_ptr));
  CUDA_CHECK(hipFree(d_col_indices));
  CUDA_CHECK(hipFree(d_dist));
  CUDA_CHECK(hipFree(d_prev_frontier));
  CUDA_CHECK(hipFree(d_prev_frontier_num));
  CUDA_CHECK(hipFree(d_curr_frontier));
  CUDA_CHECK(hipFree(d_curr_frontier_num));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}