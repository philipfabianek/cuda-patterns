#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <random>
#include <chrono>
#include <queue>
#include "graph/graph.h"

#define CUDA_CHECK(err)                                                                          \
  {                                                                                              \
    if (err != hipSuccess)                                                                      \
    {                                                                                            \
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }

constexpr int num_vertices = 128 * 1024;
constexpr int num_edges = 1024 * 1024;

constexpr int threads_per_block = 512;
constexpr int blocks_per_grid = (num_vertices + threads_per_block - 1) / threads_per_block;

/*
 * Performs BFS for a given level.
 - Finds vertices with distance equal to <prev_level> and iterates over their neighbors.
 - If the neighbor is unvisited, sets its distance to the current level (<prev_level> + 1).
 - Each thread is responsible for one vertex, one grid is launched per level.
 */
__global__ void bfs_kernel(const int *row_ptr, const int *col_indices, int *dist, int *changed, const int prev_level)
{
  int u = blockIdx.x * blockDim.x + threadIdx.x;

  if (u < num_vertices && dist[u] == prev_level)
  {
    for (int i = row_ptr[u]; i < row_ptr[u + 1]; ++i)
    {
      int v = col_indices[i];

      if (dist[v] == -1)
      {
        dist[v] = prev_level + 1;
        *changed = 1;

        // // In this case, this is not necessary,
        // // it actually leads to inferior performance
        // int old_dist = atomicCAS(&dist[v], -1, prev_level + 1);
        // if (old_dist == -1)
        // {
        //   // Mark that a change was made in this iteration
        //   *changed = 1;
        // }
      }
    }
  }
}

int main()
{
  // Create random number generator and random distribution
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);

  // Define host graph by first constructing a spanning tree
  // then adding random edges up to <num_edges>
  std::vector<Edge> edges = generate_random_spanning_tree(num_vertices, generator);
  add_random_edges(edges, num_vertices, num_edges, generator);

  // Convert the graph to the CSR format
  CSRGraph graph = convert_to_csr(edges, num_vertices);

  // Allocate memory for host output vector and initialize to -1,
  // set distance for the source node to 0
  std::vector<int> h_dist(num_vertices);
  std::fill(h_dist.begin(), h_dist.end(), -1);
  int source_vertex = 0;
  h_dist[source_vertex] = 0;

  // Prepare device variables
  size_t row_ptr_memsize = graph.row_ptr.size() * sizeof(int);
  size_t col_indices_memsize = graph.col_indices.size() * sizeof(int);
  size_t dist_memsize = graph.num_vertices * sizeof(int);
  size_t changed_memsize = sizeof(int);

  int *d_row_ptr, *d_col_indices, *d_dist, *d_changed;

  CUDA_CHECK(hipMalloc((void **)&d_row_ptr, row_ptr_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_col_indices, col_indices_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_dist, dist_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_changed, changed_memsize));

  // Copy data from host to device
  CUDA_CHECK(hipMemcpy(d_row_ptr, graph.row_ptr.data(), row_ptr_memsize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_col_indices, graph.col_indices.data(), col_indices_memsize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_dist, h_dist.data(), dist_memsize, hipMemcpyHostToDevice));

  // Create events for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  // Run BFS and compute distances on GPU
  int prev_level = 0;
  while (true)
  {
    int h_changed = 0;
    CUDA_CHECK(hipMemcpy(d_changed, &h_changed, changed_memsize, hipMemcpyHostToDevice));

    bfs_kernel<<<blocks_per_grid, threads_per_block>>>(d_row_ptr, d_col_indices, d_dist, d_changed, prev_level);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(&h_changed, d_changed, changed_memsize, hipMemcpyDeviceToHost));
    if (h_changed == 0)
    {
      break;
    }
    prev_level++;
  }

  // Record the end time and synchronize
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  // Calculate elapsed time
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Kernel execution time: %f ms (use NCU for a more precise measurement!)\n", milliseconds);

  // Copy data from device to host
  CUDA_CHECK(hipMemcpy(h_dist.data(), d_dist, dist_memsize, hipMemcpyDeviceToHost));

  // Verify the result and measure CPU execution time
  auto cpu_start = std::chrono::high_resolution_clock::now();
  if (verify_bfs(graph, h_dist, source_vertex) != 0)
  {
    printf("Verification failed\n");
  }
  else
  {
    printf("All values match\n");
  }
  auto cpu_end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double, std::milli> cpu_time = cpu_end - cpu_start;
  printf("CPU execution time: %f milliseconds\n", cpu_time.count());

  // Free memory and destroy events
  CUDA_CHECK(hipFree(d_row_ptr));
  CUDA_CHECK(hipFree(d_col_indices));
  CUDA_CHECK(hipFree(d_dist));
  CUDA_CHECK(hipFree(d_changed));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}