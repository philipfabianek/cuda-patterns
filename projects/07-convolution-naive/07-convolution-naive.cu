
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <chrono>

#define CUDA_CHECK(err)                                                                          \
  {                                                                                              \
    if (err != hipSuccess)                                                                      \
    {                                                                                            \
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }

constexpr bool random_initialization = false;
constexpr int matrix_rows = 2048;
constexpr int matrix_cols = 2048;
constexpr int filter_radius_x = 2;
constexpr int filter_radius_y = 2;
constexpr int filter_rows = (2 * filter_radius_y + 1);
constexpr int filter_cols = (2 * filter_radius_x + 1);
constexpr int samples_to_check = 10000;

constexpr dim3 threads_per_block(16, 16);
constexpr int blocks_x = (matrix_cols + threads_per_block.x - 1) / threads_per_block.x;
constexpr int blocks_y = (matrix_rows + threads_per_block.y - 1) / threads_per_block.y;
constexpr dim3 num_blocks(blocks_x, blocks_y);

__global__ void naive_convolution(float *d_A, float *d_B, float *d_F)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < matrix_rows && col < matrix_cols)
  {
    float sum = 0;

    for (int i = -filter_radius_y; i < filter_radius_y + 1; i++)
    {
      for (int j = -filter_radius_x; j < filter_radius_x + 1; j++)
      {
        if (row + i >= 0 && row + i < matrix_rows && col + j >= 0 && col + j < matrix_cols)
        {
          sum += d_A[(row + i) * matrix_cols + (col + j)] * d_F[(filter_radius_y + i) * filter_cols + (filter_radius_x + j)];
        }
      }
    }

    d_B[row * matrix_cols + col] = sum;
  }
}

int verify_convolution(float *h_A, float *h_B, float *h_F)
{
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);

  std::uniform_int_distribution<> row_dist(0, matrix_rows - 1);
  std::uniform_int_distribution<> col_dist(0, matrix_cols - 1);

  for (int s = 0; s < samples_to_check; s++)
  {
    int i = row_dist(generator);
    int j = col_dist(generator);

    float target_value = h_B[i * matrix_cols + j];
    float expected_value = 0.0f;

    for (int k = -filter_radius_y; k < filter_radius_y + 1; k++)
    {
      for (int l = -filter_radius_x; l < filter_radius_x + 1; l++)
      {
        if (i + k >= 0 && i + k < matrix_rows && j + l >= 0 && j + l < matrix_cols)
        {
          expected_value += h_A[(i + k) * matrix_cols + (j + l)] * h_F[(filter_radius_y + k) * filter_cols + (filter_radius_x + l)];
        }
      }
    }

    if (fabs(target_value - expected_value) > 1e-5)
    {
      printf("Mismatch (%d, %d): expected %f, got %f\n", i, j, expected_value, target_value);
      return 1;
    }
  }

  return 0;
}

int main()
{
  // Create random number generator and random distribution
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);
  std::uniform_real_distribution<float> distribution(-0.5f, 0.5f);

  // Define matrix A with values from a random distribution
  size_t A_memsize = matrix_rows * matrix_cols * sizeof(float);
  std::vector<float> h_A(matrix_rows * matrix_cols);

  for (int i = 0; i < matrix_rows; i++)
  {
    for (int j = 0; j < matrix_cols; j++)
    {
      if (random_initialization)
      {
        h_A[i * matrix_cols + j] = distribution(generator);
      }
      else
      {
        h_A[i * matrix_cols + j] = 1.0f;
      }
    }
  }

  // Define filter F with values from a random distribution
  size_t F_memsize = filter_rows * filter_cols * sizeof(float);
  std::vector<float> h_F(filter_rows * filter_cols);

  for (int i = 0; i < filter_rows; i++)
  {
    for (int j = 0; j < filter_cols; j++)
    {
      if (random_initialization)
        h_F[i * filter_cols + j] = distribution(generator);
      else
      {
        h_F[i * filter_cols + j] = 1.0f;
      }
    }
  }

  // Allocate memory for matrix B which will be the result of the convolution
  // (formally cross-correlation) of the matrix A with the filter F
  // (it will have same size as A, zeros will be used as padding)
  size_t B_memsize = A_memsize;
  std::vector<float> h_B(matrix_rows * matrix_cols);

  // Prepare device variables for the matrices and the filter
  float *d_A, *d_B, *d_F;
  CUDA_CHECK(hipMalloc((void **)&d_A, A_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_B, B_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_F, F_memsize));

  // Move data from host to device
  CUDA_CHECK(hipMemcpy(d_A, h_A.data(), A_memsize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_F, h_F.data(), F_memsize, hipMemcpyHostToDevice));

  // Create events for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  // Perform convolution on GPU
  naive_convolution<<<num_blocks, threads_per_block>>>(d_A, d_B, d_F);
  CUDA_CHECK(hipGetLastError());

  // Record the end time and synchronize
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  // Calculate elapsed time
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Kernel execution time: %f ms (use NCU for a more precise measurement!)\n", milliseconds);

  // Move data from device to host
  CUDA_CHECK(hipMemcpy(h_B.data(), d_B, B_memsize, hipMemcpyDeviceToHost));

  // Check values
  if (verify_convolution(h_A.data(), h_B.data(), h_F.data()) != 0)
  {
    return 1;
  }
  printf("All (sampled) values match\n");

  // Free memory and destroy events
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_F));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}