
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <random>
#include <chrono>

#define CUDA_CHECK(err)                                                                          \
  {                                                                                              \
    if (err != hipSuccess)                                                                      \
    {                                                                                            \
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }

constexpr int N = 512 * 512 * 512;
constexpr char initial_char = 'A';
constexpr int num_bins = 64;
// Cannot be too large due because of signed char
// ranging from -128 to 127 (A is 65)
constexpr int num_unique_chars = 50;

// Using 512 threads per block leads to the best performance on my GPU,
// it is slightly better than 256 threads per block
// and significantly better than 1024 threads per block
constexpr int threads_per_block = 512;
constexpr int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

/*
 * Computes a histogram of character frequencies.
 * - Each thread processes one input character.
 * - Histograms for each block are stored in shared memory
 *   before being merged into the final histogram, this reduces the number
 *   of overlapping atomic operations and significantly improves the performance.
 * - The performance increase compared to the previous privatized version
 *   is also very significant.
 */
__global__ void histogram_kernel(const char *d_input_string, unsigned int *d_histogram)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ unsigned int shared_histogram[num_bins];

  for (int i = threadIdx.x; i < num_bins; i += blockDim.x)
  {
    shared_histogram[i] = 0;
  }

  __syncthreads();

  // Compute histogram for the current block
  if (idx < N)
  {
    char current_char = d_input_string[idx];
    unsigned int destination_bin = (current_char - initial_char) * num_bins / num_unique_chars;
    atomicAdd(&shared_histogram[destination_bin], 1);
  }

  __syncthreads();

  // Merge the shared histogram into the final histogram,
  // using this loop handles cases where the number of bins
  // would be greater than the number of threads
  for (int bin = threadIdx.x; bin < num_bins; bin += blockDim.x)
  {
    atomicAdd(&d_histogram[bin], shared_histogram[bin]);
  }
}

int verify_histogram(const char *h_input_string, const unsigned int *h_histogram)
{
  unsigned int target_histogram[num_bins] = {0};

  for (int i = 0; i < N; i++)
  {
    char current_char = h_input_string[i];
    unsigned int bin_index = (current_char - initial_char) * num_bins / num_unique_chars;
    target_histogram[bin_index]++;
  }

  for (int i = 0; i < num_bins; i++)
  {
    if (target_histogram[i] != h_histogram[i])
    {
      printf("Mismatch at bin %d: expected %u, got %u\n", i, target_histogram[i], h_histogram[i]);
      return 1;
    }
  }

  return 0;
}

int main()
{
  // Define input string
  size_t input_memsize = N * sizeof(char);
  std::vector<char> h_input_string(N);

  // Make 90% characters the same to make the optimization more challenging,
  // use all characters for the last 10%
  for (int i = 0; i < N; i++)
  {
    if (i < 0.9 * N)
    {
      h_input_string[i] = initial_char;
    }
    else
    {
      h_input_string[i] = initial_char + (i % (num_unique_chars - 1)) + 1;
    }
  }

  // Allocate memory for the host histogram result
  size_t histogram_memsize = num_bins * sizeof(unsigned int);
  std::vector<unsigned int> h_histogram(num_bins);

  // Prepare device variables
  char *d_input_string;
  unsigned int *d_histogram;
  CUDA_CHECK(hipMalloc((void **)&d_input_string, input_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_histogram, histogram_memsize));

  // Move data from host to device
  CUDA_CHECK(hipMemcpy(d_input_string, h_input_string.data(), input_memsize, hipMemcpyHostToDevice));

  // Initialize the histogram memory on the device to all zeros
  CUDA_CHECK(hipMemset(d_histogram, 0, histogram_memsize));

  // Create events for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  // Perform histogram calculation on GPU
  histogram_kernel<<<blocks_per_grid, threads_per_block>>>(d_input_string, d_histogram);
  CUDA_CHECK(hipGetLastError());

  // Record the end time and synchronize
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  // Calculate elapsed time
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Kernel execution time: %f ms (use NCU for a more precise measurement!)\n", milliseconds);

  // Move data from device to host
  CUDA_CHECK(hipMemcpy(h_histogram.data(), d_histogram, histogram_memsize, hipMemcpyDeviceToHost));

  // Check values
  printf("Verifying histogram...\n");
  if (verify_histogram(h_input_string.data(), h_histogram.data()) != 0)
  {
    return 1;
  }
  printf("All values match\n");

  // Free device memory
  CUDA_CHECK(hipFree(d_input_string));
  CUDA_CHECK(hipFree(d_histogram));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}