
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <random>
#include <chrono>

#define CUDA_CHECK(err)                                                                          \
  {                                                                                              \
    if (err != hipSuccess)                                                                      \
    {                                                                                            \
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }

constexpr int matrix_rows = 4096;
constexpr int matrix_cols = 4096;
constexpr float sparsity = 0.01f;
constexpr int num_non_zeros = matrix_rows * matrix_cols * sparsity;

constexpr int threads_per_block = 256;
constexpr int blocks_per_grid = (num_non_zeros + threads_per_block - 1) / threads_per_block;

/*
 * Performs Sparse Matrix-Vector multiplication (SpMV) using the coordinate (COO) format.
 * - Each thread computes the product of one non-zero element of the matrix.
 * - atomicAdd is used to prevent race conditions.
 */
__global__ void spmv_coo_kernel(const float *value, const int *rowIdx, const int *colIdx,
                                const float *x, float *y)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_non_zeros)
  {
    float v = value[idx];
    int r = rowIdx[idx];
    int c = colIdx[idx];
    atomicAdd(&y[r], v * x[c]);
  }
}

/*
 * Verifies the result of the SpMV computation from the GPU
 * by comparing it against a CPU-based computation.
 */
int verify_spmv(const float *h_value, const int *h_rowIdx, const int *h_colIdx, const float *h_x,
                const float *h_y_gpu, int n_non_zeros, int n_rows)
{
  std::vector<float> h_y_cpu(n_rows, 0.0f);

  for (int i = 0; i < n_non_zeros; ++i)
  {
    h_y_cpu[h_rowIdx[i]] += h_value[i] * h_x[h_colIdx[i]];
  }

  int status = 0;
  for (int i = 0; i < n_rows; ++i)
  {
    if (fabs(h_y_cpu[i] - h_y_gpu[i]) > 1e-3)
    {
      printf("Mismatch at row %d: expected %f, got %f\n", i, h_y_cpu[i], h_y_gpu[i]);
      status = 1;
      break;
    }
  }

  return status;
}

int main()
{
  // Create random number generator and random distribution
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);
  std::uniform_real_distribution<float> value_dist(-1.0f, 1.0f);
  std::uniform_int_distribution<int> row_dist(0, matrix_rows - 1);
  std::uniform_int_distribution<int> col_dist(0, matrix_cols - 1);

  // Define host sparse matrix in COO format with values from the random distribution
  std::vector<float> h_value(num_non_zeros);
  std::vector<int> h_rowIdx(num_non_zeros);
  std::vector<int> h_colIdx(num_non_zeros);

  for (int i = 0; i < num_non_zeros; ++i)
  {
    h_value[i] = value_dist(generator);
    h_rowIdx[i] = row_dist(generator);
    h_colIdx[i] = col_dist(generator);
  }

  // Define host input vector
  std::vector<float> h_x(matrix_cols);
  for (int i = 0; i < matrix_cols; ++i)
  {
    h_x[i] = value_dist(generator);
  }

  // Allocate memory for host output vector
  std::vector<float> h_y(matrix_rows);

  // Prepare device variables
  size_t value_memsize = num_non_zeros * sizeof(float);
  size_t idx_memsize = num_non_zeros * sizeof(int);
  size_t x_memsize = matrix_cols * sizeof(float);
  size_t y_memsize = matrix_rows * sizeof(float);

  float *d_value, *d_x, *d_y;
  int *d_rowIdx, *d_colIdx;

  CUDA_CHECK(hipMalloc((void **)&d_value, value_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_rowIdx, idx_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_colIdx, idx_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_x, x_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_y, y_memsize));

  // Copy data from host to device
  CUDA_CHECK(hipMemcpy(d_value, h_value.data(), value_memsize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_rowIdx, h_rowIdx.data(), idx_memsize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_colIdx, h_colIdx.data(), idx_memsize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_x, h_x.data(), x_memsize, hipMemcpyHostToDevice));

  // Ensure output vector is initialized to zero
  CUDA_CHECK(hipMemset(d_y, 0, y_memsize));

  // Create events for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  // Compute sparse matrix-vector multiplication on GPU
  spmv_coo_kernel<<<blocks_per_grid, threads_per_block>>>(d_value, d_rowIdx, d_colIdx, d_x, d_y);
  CUDA_CHECK(hipGetLastError());

  // Record the end time and synchronize
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  // Calculate elapsed time
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Kernel execution time: %f ms (use NCU for a more precise measurement!)\n", milliseconds);

  // Copy data from device to host
  CUDA_CHECK(hipMemcpy(h_y.data(), d_y, y_memsize, hipMemcpyDeviceToHost));

  // Verify the result
  if (verify_spmv(h_value.data(), h_rowIdx.data(), h_colIdx.data(), h_x.data(), h_y.data(), num_non_zeros, matrix_rows) != 0)
  {
    printf("Verification failed\n");
  }
  else
  {
    printf("All values match\n");
  }

  // Free memory and destroy events
  CUDA_CHECK(hipFree(d_value));
  CUDA_CHECK(hipFree(d_rowIdx));
  CUDA_CHECK(hipFree(d_colIdx));
  CUDA_CHECK(hipFree(d_x));
  CUDA_CHECK(hipFree(d_y));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}