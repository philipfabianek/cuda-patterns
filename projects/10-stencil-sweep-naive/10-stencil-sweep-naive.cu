
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <random>
#include <chrono>

#define CUDA_CHECK(err)                                                                          \
  {                                                                                              \
    if (err != hipSuccess)                                                                      \
    {                                                                                            \
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); \
      exit(EXIT_FAILURE);                                                                        \
    }                                                                                            \
  }

constexpr bool random_initialization = false;
constexpr int N = 512;
constexpr int block_size = 8;
constexpr int samples_to_check = 100000;

constexpr dim3 threads_per_block(block_size, block_size, block_size);
constexpr int blocks_x = (N + block_size - 1) / block_size;
constexpr int blocks_y = (N + block_size - 1) / block_size;
constexpr int blocks_z = (N + block_size - 1) / block_size;
constexpr dim3 num_blocks(blocks_x, blocks_y, blocks_z);

/*
 * Performs a 3D stencil sweep on a 3D tensor using a 7-point stencil.
 * - Each thread computes one element of the output tensor.
 * - Boundary elements are set to zero.
 */
__global__ void stencil_kernel(float *d_A, float *d_B)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (x >= 1 &&
      x < N - 1 &&
      y >= 1 &&
      y < N - 1 &&
      z >= 1 &&
      z < N - 1)
  {
    // Approximate inner values using a 7-point stencil
    float sum = d_A[z * N * N + y * N + x] +
                d_A[(z - 1) * N * N + y * N + x] +
                d_A[(z + 1) * N * N + y * N + x] +
                d_A[z * N * N + (y - 1) * N + x] +
                d_A[z * N * N + (y + 1) * N + x] +
                d_A[z * N * N + y * N + (x - 1)] +
                d_A[z * N * N + y * N + (x + 1)];
    d_B[z * N * N + y * N + x] = sum;
  }
  else
  {
    // Set boundary values to zero
    if (x < N && y < N && z < N)
    {
      d_B[z * N * N + y * N + x] = 0.0f;
    }
  }
}

int verify_stencil_sweep(float *h_A, float *h_B)
{
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);

  std::uniform_int_distribution<> x_dist(0, N - 1);
  std::uniform_int_distribution<> y_dist(0, N - 1);
  std::uniform_int_distribution<> z_dist(0, N - 1);

  for (int s = 0; s < samples_to_check; s++)
  {
    int i = x_dist(generator);
    int j = y_dist(generator);
    int k = z_dist(generator);

    float target_value = h_B[k * N * N + j * N + i];
    float expected_value = 0.0f;

    if (i >= 1 && i < N - 1 &&
        j >= 1 && j < N - 1 &&
        k >= 1 && k < N - 1)
    {
      expected_value = h_A[k * N * N + j * N + i] +
                       h_A[(k - 1) * N * N + j * N + i] +
                       h_A[(k + 1) * N * N + j * N + i] +
                       h_A[k * N * N + (j - 1) * N + i] +
                       h_A[k * N * N + (j + 1) * N + i] +
                       h_A[k * N * N + j * N + (i - 1)] +
                       h_A[k * N * N + j * N + (i + 1)];
    }

    if (fabs(target_value - expected_value) > 1e-5)
    {
      printf("Mismatch (%d, %d): expected %f, got %f\n", i, j, expected_value, target_value);
      return 1;
    }
  }

  return 0;
}

int main()
{
  // Create random number generator and random distribution
  unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
  std::default_random_engine generator(seed);
  std::uniform_real_distribution<float> distribution(-0.5f, 0.5f);

  // Define 3D data tensor
  size_t A_memsize = N * N * N * sizeof(float);
  std::vector<float> h_A(N * N * N);

  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      for (int k = 0; k < N; k++)
      {
        if (random_initialization)
        {
          h_A[i * N * N + j * N + k] = distribution(generator);
        }
        else
        {
          h_A[i * N * N + j * N + k] = 1.0f;
        }
      }
    }
  }

  // Allocate memory for tensor B which will be the result
  // of applying the stencil kernel to the tensor A
  size_t B_memsize = A_memsize;
  std::vector<float> h_B(N * N * N);

  // Prepare device variables for the tensors
  float *d_A, *d_B;
  CUDA_CHECK(hipMalloc((void **)&d_A, A_memsize));
  CUDA_CHECK(hipMalloc((void **)&d_B, B_memsize));

  // Move data from host to device
  CUDA_CHECK(hipMemcpy(d_A, h_A.data(), A_memsize, hipMemcpyHostToDevice));

  // Create events for timing
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  CUDA_CHECK(hipEventRecord(start));

  // Perform stencil sweep on GPU
  stencil_kernel<<<num_blocks, threads_per_block>>>(d_A, d_B);
  CUDA_CHECK(hipGetLastError());

  // Record the end time and synchronize
  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  // Calculate elapsed time
  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
  printf("Kernel execution time: %f ms (use NCU for a more precise measurement!)\n", milliseconds);

  // Move data from device to host
  CUDA_CHECK(hipMemcpy(h_B.data(), d_B, B_memsize, hipMemcpyDeviceToHost));

  // Check values
  if (verify_stencil_sweep(h_A.data(), h_B.data()) != 0)
  {
    return 1;
  }
  printf("All (sampled) values match\n");

  // Free memory and destroy events
  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));

  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));

  return 0;
}